
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define SIZE (10240*10240)
#define THREADS_PER_BLOCK 512
#define NSTREAMS 4

__global__ void mat_mul(float* dA, float* dB, float* dC, int N){
    int idx = blockIdx.y*blockDim.y+threadIdx.y;
    int idy = blockIdx.x*blockDim.x+threadIdx.x;

    float blockSum = 0;

    if (idx < N && idy < N){
        for (int i = 0; i < N; i++)
            blockSum += dA[idx*N + i] * dB[i*N + idy];
    }
    dC[idx*N + idy] = blockSum;
}

__global__ void mat_mul_async(float* dA, float* dB, float* dC, int N, int id){
    int idx = blockIdx.y*blockDim.y+threadIdx.y;
    int idy = blockIdx.x*blockDim.x+threadIdx.x;

    float blockSum = 0;
    
    int STREAM_SIZE = N/NSTREAMS; 
    
    if (idx < N && idy < N && STREAM_SIZE%idx == STREAM_SIZE%idy == id){
        for (int i = 0; i < N; i++)
            blockSum += dA[idx*N + i] * dB[idy*N + i];
    }
    
    if (idx < N && idy < N && STREAM_SIZE%idx != STREAM_SIZE%idy && STREAM_SIZE%idx == id){
        for (int i = 0; i < N; i++)
            blockSum += dA[idx*N + i] * dB[idy*N + i];
    }
    
    dC[idx*N + idy] = blockSum;
}


int main (int argc, char *argv[])
{
    int N = int(sqrtf(SIZE));
    float *hA, *hB, *hC, *dA, *dB, *dC;

    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Global memory
    hA = (float*) malloc (SIZE * sizeof(float));
    hB = (float*) malloc (SIZE * sizeof(float));
    hC = (float*) malloc (SIZE * sizeof(float));
    hipMalloc((void**) &dA, SIZE * sizeof(float));
    hipMalloc((void**) &dB, SIZE * sizeof(float));
    hipMalloc((void**) &dC, SIZE * sizeof(float));

    hipEventRecord(start, 0);
    
    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);
    
    if (SIZE > THREADS_PER_BLOCK){
        threadsPerBlock.x = THREADS_PER_BLOCK;
        threadsPerBlock.y = THREADS_PER_BLOCK;
        blocksPerGrid.x = N/THREADS_PER_BLOCK;
        blocksPerGrid.y = N/THREADS_PER_BLOCK;
    }

    hipMemcpy(dA, hA, sizeof(float)*SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, sizeof(float)*SIZE, hipMemcpyHostToDevice);

    mat_mul<<< blocksPerGrid, threadsPerBlock >>>(dA, dB, dC, N);
    hipMemcpy(hC, dC, N * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time for CUDA GPU (global memory):  %3.1f ms \n", time);

    free(hA);
    free(hB);
    free(hC);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    
    // Pinned memory
    hipHostMalloc((void**) &hA, SIZE * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void**) &hB, SIZE * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void**) &hC, SIZE * sizeof(float), hipHostMallocDefault);
    hipMalloc( (void**) &dA, SIZE * sizeof(float));
    hipMalloc((void**) &dB, SIZE * sizeof(float));
    hipMalloc((void**) &dC, SIZE * sizeof(float));
    
    hipEventRecord(start, 0);
    
    hipMemcpy(dA, hA, sizeof(float)*SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, sizeof(float)*SIZE, hipMemcpyHostToDevice);

    mat_mul<<< blocksPerGrid, threadsPerBlock >>>(dA, dB, dC, N);
    hipMemcpy(hC, dC, N * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time for CUDA GPU (pinned memory):  %3.1f ms \n", time);

    hipHostFree(hA);
    hipHostFree(hB);
    hipHostFree(hC);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    
    // Pinned memory and Async    
    hipHostMalloc((void**) &hA, SIZE * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void**) &hB, SIZE * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void**) &hC, SIZE * sizeof(float), hipHostMallocDefault);
    hipMalloc( (void**) &dA, SIZE * sizeof(float));
    hipMalloc((void**) &dB, SIZE * sizeof(float));
    hipMalloc((void**) &dC, SIZE * sizeof(float));
    
    int STREAM_SIZE = int(SIZE/NSTREAMS);
    hipStream_t stream[NSTREAMS];
    
    for (int i = 0; i < NSTREAMS; i++){
        hipStreamCreate(&stream[i]);
    }
    
    hipEventRecord(start, 0);
    
    for (int i = 0; i < NSTREAMS; i++){
        int offset = i*STREAM_SIZE;
        hipMemcpyAsync(&dA[offset], &hA[offset], sizeof(float)*STREAM_SIZE, hipMemcpyHostToDevice, stream[i]);
    }
    
    for (int i = 0; i < NSTREAMS; i++){
        int offset = i*STREAM_SIZE;
        hipMemcpyAsync(&dB[offset], &hB[offset], sizeof(float)*STREAM_SIZE, hipMemcpyHostToDevice, stream[i]);
    }
    
    hipDeviceSynchronize();
    mat_mul<<< blocksPerGrid, threadsPerBlock>>>(dA, dB, dC, N);
    
    for (int i = 0; i < NSTREAMS; i++){
        int offset = i*STREAM_SIZE;
        hipMemcpyAsync(&hC[offset], &dC[offset], sizeof(float)*STREAM_SIZE, hipMemcpyDeviceToHost, stream[i]);
    }
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time for CUDA GPU (pinned memory and async):  %3.1f ms \n", time);

    hipHostFree(hA);
    hipHostFree(hB);
    hipHostFree(hC);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    for (int i = 0; i < NSTREAMS; i++){
        hipStreamDestroy(stream[i]);
    }

    return 0;
}

