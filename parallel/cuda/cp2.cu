
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <iostream>

#define SIZE (4*4)
#define THREADS_PER_BLOCK 16
#define NSTREAMS 4
#define TILE_WIDTH 16

__global__ void mat_mul(float* dA, float* dB, float* dC, int N){
    int idx = blockIdx.y*blockDim.y+threadIdx.y;
    int idy = blockIdx.x*blockDim.x+threadIdx.x;

    float blockSum = 0;

    if (idx < N && idy < N){
        for (int i = 0; i < N; i++)
            blockSum += dA[idx*N + i] * dB[i*N + idy];
    }
    dC[idx*N + idy] = blockSum;
}

__global__ void mat_mul_shared(float* dA, float* dB, float* dC, int N){
    
//     __shared__ float dM[TILE_WIDTH][TILE_WIDTH];
//     __shared__ float dN[TILE_WIDTH][TILE_WIDTH];
//     
//      int tx = threadIdx.x, ty = threadIdx.y;
//      int idx = blockIdx.y * TILE_WIDTH + ty;
//      int idy = blockIdx.x * TILE_WIDTH + tx;
//     float Pvalue = 0;
// 
//     for (int m = 0; m < (numAColumns-1)/TILE_WIDTH+1; ++m) {
//        if (Row < numARows && m*TILE_WIDTH+tx < numAColumns)
//           ds_M[ty][tx] = A[Row*numAColumns + m*TILE_WIDTH+tx];
//        else
//           ds_M[ty][tx] = 0;
//        if (Col < numBColumns && m*TILE_WIDTH+ty < numBRows)
//           ds_N[ty][tx] = B[(m*TILE_WIDTH+ty)*numBColumns+Col];
//        else
//           ds_N[ty][tx] = 0;
// 
//        __syncthreads();
//        for (int k = 0; k < TILE_WIDTH; ++k)
//           Pvalue += ds_M[ty][k] * ds_N[k][tx];
//        __syncthreads();
//     }
//     if (Row < numCRows && Col < numCColumns)
//        C[Row*numCColumns+Col] = Pvalue;
}

__global__ void mat_mul_async1(float* dA, float* dB, float* dC, int N, int id){
    int idx = blockIdx.y*blockDim.y+threadIdx.y;
    int idy = blockIdx.x*blockDim.x+threadIdx.x;

    float blockSum = 0;
    
    int STREAM_SIZE = N/NSTREAMS; 
    
    if (idx < N && idy < N && STREAM_SIZE%idx == STREAM_SIZE%idy == id){
        for (int i = 0; i < N; i++)
            blockSum += 2;//dA[idx*N + i] * dB[idy*N + i];
    }
    
    dC[idy*N + idx] = 2;
}

__global__ void mat_mul_async2(float* dA, float* dB, float* dC, int N, int id){
    int idx = blockIdx.y*blockDim.y+threadIdx.y;
    int idy = blockIdx.x*blockDim.x+threadIdx.x;

    float blockSum = 0;
    
    int STREAM_SIZE = N/NSTREAMS; 
    
    if (idx < N && idy < N && STREAM_SIZE%idx != STREAM_SIZE%idy && STREAM_SIZE%idx == id){
        for (int i = 0; i < N; i++)
            blockSum += dA[idx*N + i] * dB[idy*N + i];
    }
    
    dC[idy*N + idx] = blockSum;
}


int main (int argc, char *argv[])
{
    int N = int(sqrtf(SIZE));
    float *hA, *hB, *hC, *dA, *dB, *dC;

    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Global memory
    hA = (float*) malloc (SIZE * sizeof(float));
    hB = (float*) malloc (SIZE * sizeof(float));
    hC = (float*) malloc (SIZE * sizeof(float));
    hipMalloc((void**) &dA, SIZE * sizeof(float));
    hipMalloc((void**) &dB, SIZE * sizeof(float));
    hipMalloc((void**) &dC, SIZE * sizeof(float));

    hipEventRecord(start, 0);
    
    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);
    
    if (SIZE > THREADS_PER_BLOCK){
        threadsPerBlock.x = THREADS_PER_BLOCK;
        threadsPerBlock.y = THREADS_PER_BLOCK;
        blocksPerGrid.x = N/THREADS_PER_BLOCK;
        blocksPerGrid.y = N/THREADS_PER_BLOCK;
    }

    hipMemcpy(dA, hA, sizeof(float)*SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, sizeof(float)*SIZE, hipMemcpyHostToDevice);

    mat_mul<<< blocksPerGrid, threadsPerBlock >>>(dA, dB, dC, N);
    hipMemcpy(hC, dC, SIZE * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time for CUDA GPU (global memory):  %3.1f ms \n", time);
    
    free(hA);
    free(hB);
    free(hC);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    
    // Pinned memory
    hipHostMalloc((void**) &hA, SIZE * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void**) &hB, SIZE * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void**) &hC, SIZE * sizeof(float), hipHostMallocDefault);
    hipMalloc( (void**) &dA, SIZE * sizeof(float));
    hipMalloc((void**) &dB, SIZE * sizeof(float));
    hipMalloc((void**) &dC, SIZE * sizeof(float));
    
    hipEventRecord(start, 0);
    
    hipMemcpy(dA, hA, sizeof(float)*SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, sizeof(float)*SIZE, hipMemcpyHostToDevice);

    mat_mul<<< blocksPerGrid, threadsPerBlock >>>(dA, dB, dC, N);
    hipMemcpy(hC, dC, SIZE * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time for CUDA GPU (pinned memory):  %3.1f ms \n", time);
    
    hipHostFree(hA);
    hipHostFree(hB);
    hipHostFree(hC);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    
    // Pinned memory and Async    
    hipHostMalloc((void**) &hA, SIZE * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void**) &hB, SIZE * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void**) &hC, SIZE * sizeof(float), hipHostMallocDefault);
    hipMalloc( (void**) &dA, SIZE * sizeof(float));
    hipMalloc((void**) &dB, SIZE * sizeof(float));
    hipMalloc((void**) &dC, SIZE * sizeof(float));
    
    int STREAM_SIZE = int(SIZE/NSTREAMS);
    hipStream_t stream[NSTREAMS];
    
    for (int i = 0; i < NSTREAMS; i++){
        hipStreamCreate(&stream[i]);
    }
    
    hipEventRecord(start, 0);
    
    for (int i = 0; i < NSTREAMS; i++){
        int offset = i*STREAM_SIZE;
        hipMemcpyAsync(&dA[offset], &hA[offset], sizeof(float)*STREAM_SIZE, hipMemcpyHostToDevice, stream[i]);
        hipMemcpyAsync(&dB[offset], &hB[offset], sizeof(float)*STREAM_SIZE, hipMemcpyHostToDevice, stream[i]);
        mat_mul_async1<<< blocksPerGrid, threadsPerBlock, 0, stream[i] >>>(&dA[offset], &dB[offset], &dC[offset], N/NSTREAMS, i);
        mat_mul_async2<<< blocksPerGrid, threadsPerBlock, 0, stream[i] >>>(dA, dB, dC, N, i);
        hipMemcpyAsync(&hC[offset], &dC[offset], sizeof(float)*STREAM_SIZE, hipMemcpyDeviceToHost, stream[i]);
    }
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time for CUDA GPU (pinned memory and async):  %3.1f ms \n", time);
    
    for(int i = 0; i < SIZE; i++)
        std::cout << hC[i] << std::endl;
    
    hipHostFree(hA);
    hipHostFree(hB);
    hipHostFree(hC);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    for (int i = 0; i < NSTREAMS; i++){
        hipStreamDestroy(stream[i]);
    }
    
    auto res = hipGetLastError();
    if(res != hipSuccess){
        printf("%s\n", hipGetErrorString(res));
    }
    else{
        printf("OK \n");
    }
    
    return 0;
}

