
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 100*1024*1024

__global__ void fx(float* dA){
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	dA[idx] = sinf(sqrtf(2.0*3.14*(float)idx/(float)N));
}

void fx_cpu(float* hA){
	for (int idx = 0; idx < N; idx++)
	    hA[idx] = sinf(sqrtf(2.0*3.14*(float)idx/(float)N));
}

int main(){
	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float *hA, *dA;

	// Global memory
	hA = (float*) malloc(sizeof(float)*N);
	hipMalloc( (void**) &dA, sizeof(float)*N);
	
	hipEventRecord(start, 0);	
	
	fx <<<N/512, 512>>> (dA);
	

	hipMemcpy(hA, dA, sizeof(float)*N, hipMemcpyDeviceToHost);
	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start,stop);
	printf("Time for GPU Global = %3.1f ms\n", time);
	hipFree(dA);
	free(hA);
	
	// Pinned memory
	hipHostMalloc((void**) &hA, sizeof(float)*N, hipHostMallocDefault);
	hipMalloc( (void**) &dA, sizeof(float)*N);
	
	hipEventRecord(start, 0);	
	
	fx <<<N/512, 512>>> (dA);
	

	hipMemcpy(hA, dA, sizeof(float)*N, hipMemcpyDeviceToHost);
	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start,stop);
	printf("Time for GPU Pinned = %3.1f ms\n", time);
	
	hipFree(dA);
	hipHostFree(hA);
	
	// UVA memory
	hipMallocManaged((void**) &hA, sizeof(float)*N);
	
	hipEventRecord(start, 0);	
	
	fx <<<N/512, 512>>> (hA);
	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start,stop);
	printf("Time for GPU UVA = %3.1f ms\n", time);
	
	hipEventRecord(start, 0);	
	
	fx_cpu (hA);
	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start,stop);
	printf("Time for CPU UVA = %3.1f ms\n", time);
    
	hipFree(hA);
	
	// Serial Method
	
	hA = (float*) malloc(sizeof(float)*N);
	hipEventRecord(start, 0);	
	
	fx_cpu(hA);
	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start,stop);
	printf("Time for CPU Serial= %3.1f ms\n", time);
	free(hA); 
	return 0;
}
